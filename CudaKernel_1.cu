#pragma once
// includes, C string library
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <cmath>
#include <hip/hip_runtime.h>

//-----------------------------------
//kernel function to update the vertex buffer
//-----------------------------------
__global__ void kernel(float4 *pos,unsigned int width,unsigned int height,float time)
{
	unsigned int x=blockIdx.x*blockDim.x+threadIdx.x;
    unsigned int y=blockIdx.y*blockDim.y+threadIdx.y;
	//calculate uv coordinates
	float u=x;
	float v=y;
	//calculate simple sine wave pattern
	float freq=0.06f;  //the smaller the less wave
	float w=sinf(u*freq+time)*cosf(v*freq+time)*2.0f;   
    //write output vertex
	u=u-(float)blockDim.x*gridDim.x/2;
	v=v-(float)blockDim.y*gridDim.y/2;
	pos[y*width+x]=make_float4(u/4,w,v/4,__int_as_float(0xff1e90ff));  //color code=lake blue ,****MY FAVOURITE NOKIA COLOR****
}

extern "C"
void D3DKernel(float4 *pos,unsigned int width,unsigned int height,float time )
{
	hipDeviceProp_t prop;
	int num=0; //cudaDevice count
	hipError_t cudaStatus=hipGetDeviceCount(&num);
	for(int i=0;i<num;i++)
		hipGetDeviceProperties(&prop,i);
	hipSetDevice(0); //set the Tesla C1060
    dim3 block(8, 8, 1);
    dim3 grid((width+block.x-1)/ block.x, (height+block.y-1) / block.y, 1);

    kernel<<<grid, block>>>(pos, width, height, time);

}